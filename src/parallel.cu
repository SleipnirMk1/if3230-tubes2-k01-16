#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "serial.c"

// Parallel odd-even sort
__global__ void parallel_odd_even(int *d_arr, int num_of_elements) {
  int idx = threadIdx.x;
  int temp;

  // Get every odd index
  idx = idx * 2 + 1;

  // Make sure no index out of range
  if (idx <= num_of_elements-2) { 
      // Check with left
      if (d_arr[idx-1] > d_arr[idx]) { 
          // Swap
          temp = d_arr[idx-1];
          d_arr[idx-1] = d_arr[idx];
          d_arr[idx] = temp;
      }
  }

  // Idk why, checking right and left in the same if statement yields wrong results
  if (idx <= num_of_elements-2) { 
      // Check with right
      if (d_arr[idx+1] < d_arr[idx]) {
          // Swap
          temp = d_arr[idx];
          d_arr[idx] = d_arr[idx+1];
          d_arr[idx+1] = temp;
      }
  }

}

// Sort that is called on main
// Usage: d_arr = array to be sorted; num_of_elements = array length
void odd_even(int *d_arr, int num_of_elements) {

  // Repeat for 1/2 length of input
  for (int i = 0; i <= num_of_elements/2; i++) {
      parallel_odd_even<<<1, num_of_elements>>>(d_arr, num_of_elements);
  }
}

// matrix convolution w/ CUDA
__global__ void convolution_cuda(Matrix *kernel, Matrix *target, Matrix *out, int *row, int *col) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int intermediate_sum = 0;
    for (int i = 0; i < kernel->row_eff; i++) {
        for (int j = 0; j < kernel->col_eff; j++) {
            intermediate_sum += kernel->mat[i][j] * target->mat[i + idx / *row][j + idx / *col];
        }
    }

    out->mat[idx / *row][idx / *col] = intermediate_sum;
}

int main() {
    
  int kernel_row, kernel_col, target_row, target_col, num_targets, conv_row, conv_col, median, floored_mean;
  int *cu_row, *cu_col, *cu_arr;
  Matrix kernel, output;
  Matrix *arr_mat, *kernel_mat, *target_mat, *output_mat;

  clock_t start = clock();

  // read kernel matrix
  scanf("%d %d", &kernel_row, &kernel_col);
  kernel = input_matrix(kernel_row, kernel_col);

  // read target matrix & data range array
  scanf("%d %d %d", &num_targets, &target_row, &target_col);
  int arr_range[num_targets];
  arr_mat = (Matrix *)malloc(num_targets * sizeof(Matrix));

  // conv matrix row & col 
  conv_row = target_row - kernel_row;
  conv_col = target_col - kernel_col;

  // create output matrix
  init_matrix(&output, conv_row, conv_col);

  // conv row & col memory allocation
	hipMalloc((void **)&cu_row, sizeof(int));
	hipMalloc((void **)&cu_col, sizeof(int));

  // sort array memory allocation
  hipMalloc((void **)&cu_arr, num_targets*sizeof(int));
    
  // matrix memory allocation
	hipMalloc((void **)&kernel_mat, sizeof(Matrix));
	hipMalloc((void **)&target_mat, sizeof(Matrix)); 
	hipMalloc((void **)&output_mat, sizeof(Matrix));

  // copy conv row & col
	hipMemcpy(cu_row, &conv_row, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_col, &conv_col, sizeof(int), hipMemcpyHostToDevice);

  // copy kernel & output matrix
  hipMemcpy(kernel_mat, &kernel, sizeof(Matrix), hipMemcpyHostToDevice);
	hipMemcpy(output_mat, &output, sizeof(Matrix), hipMemcpyHostToDevice);

  // convolution
  for (int i = 0; i < num_targets; i++) {
      arr_mat[i] = input_matrix(target_row, target_col);
      hipMemcpy(target_mat, &arr_mat[i], sizeof(Matrix), hipMemcpyHostToDevice);

      // conv process
      convolution_cuda<<<1,128>>>(kernel_mat, target_mat, output_mat, cu_row, cu_col);

      // write output
      hipError_t err = hipMemcpy(&arr_mat[i], output_mat, sizeof(Matrix), hipMemcpyDeviceToHost);
      if (err != hipSuccess) {
        printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
      }

      arr_range[i] = get_matrix_datarange(&arr_mat[i]);
  }

  // sorting odd even
  hipError_t err = hipMemcpy(cu_arr, arr_range, num_targets*sizeof(int), hipMemcpyHostToDevice);
  if(err !=hipSuccess) {
    printf("CUDA error copying to Device for sorting: %s\n", hipGetErrorString(err));
  }
  odd_even(cu_arr, num_targets);

  err = hipMemcpy(arr_range, cu_arr, num_targets*sizeof(int), hipMemcpyDeviceToHost);
  if(err !=hipSuccess) {
    printf("CUDA error copying to Host from sorting: %s\n", hipGetErrorString(err));
  }

  // print the min, max, median, and floored mean of data range array
  median = get_median(arr_range, num_targets);
  floored_mean = get_floored_mean(arr_range, num_targets);
  printf("min: %d\nmax: %d\nmedian: %d\nfloored mean: %d\n",
          arr_range[0],
          arr_range[num_targets - 1],
          median,
          floored_mean);

  // print duration
  double duration = (double)(clock() - start) / CLOCKS_PER_SEC;
  printf("Processing Time: %f\n", duration);

  // Cleanup
  hipFree(cu_row);
  hipFree(cu_col);
  hipFree(kernel_mat);
  hipFree(target_mat);
  hipFree(output_mat);

  return 0;
}